#include "hip/hip_runtime.h"
// screenshot to jpeg for linux

#include <sys/time.h>

#include "shot2jpeg.h"

#define IMG_AT(x, y, i) image->data[(((y_width) + (x)) << 2) + (i)]
#define RGBA_AT(x, y, i) rgba[(((y_width) + (x)) << 2) + (i)]
#define RGB_AT(x, y, i) rgb[((y_width) + (x)) * 3 + (i)]

xcb_image_t *take_screenshot(xcb_connection_t *conn, xcb_screen_t *screen) {
    return xcb_image_get(conn,
        screen->root,
        0, 0,
        screen->width_in_pixels, screen->height_in_pixels,
        UINT32_MAX,
        XCB_IMAGE_FORMAT_Z_PIXMAP);
}

xcb_pixmap_t image_to_pixmap(xcb_connection_t *conn, xcb_screen_t *screen, xcb_image_t *image) {
    xcb_pixmap_t pixmap = xcb_generate_id(conn);
    xcb_create_pixmap(conn, 24, pixmap, screen->root, image->width, image->height);

    xcb_gcontext_t gc = xcb_generate_id(conn);
    uint32_t pixels[2] =  {screen->black_pixel, 0xffffff};
    xcb_create_gc(conn, gc, pixmap,
        XCB_GC_FOREGROUND | XCB_GC_BACKGROUND,
        pixels);

    xcb_image_put(conn, pixmap, gc, image, 0, 0, 0);

    return pixmap;
}

void get_rgba_image_data(xcb_image_t *image, uint8_t *rgba) {
    for (int y = 0; y < image->height; y++) {
        int y_width = y*image->width;
        for (int x = 0; x < image->width; x++) {
            RGBA_AT(x, y, 0) = IMG_AT(x, y, 2); // r
            RGBA_AT(x, y, 1) = IMG_AT(x, y, 1); // g
            RGBA_AT(x, y, 2) = IMG_AT(x, y, 0); // b
            RGBA_AT(x, y, 3) = IMG_AT(x, y, 3); // a
        }
    }
}

void get_rgba_image_data2(xcb_image_t *image, uint8_t *rgba) {
    memcpy(rgba, image->data, image->size);
    for (int y = 0; y < image->height; y++) {
        int y_width = y*image->width;
        for (int x = 0; x < image->width; x++) {
            RGBA_AT(x, y, 0) = IMG_AT(x, y, 2); // r
            RGBA_AT(x, y, 2) = IMG_AT(x, y, 0); // b
        }
    }
}

__global__
void get_rgba_image_data3(int n, uint8_t *data) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride) {
        int x = i*4;
        uint8_t b = data[x + 2];
        data[x + 2] = data[x];
        data[x] = b;
    }
} 

void get_rgb_image_data(xcb_image_t *image, uint8_t *rgb) {
    for (int y = 0; y < image->height; y++) {
        int y_width = y*image->width;
        for (int x = 0; x < image->width; x++) {
            RGB_AT(x, y, 0) = IMG_AT(x, y, 2); // r
            RGB_AT(x, y, 1) = IMG_AT(x, y, 1); // g
            RGB_AT(x, y, 2) = IMG_AT(x, y, 0); // b
        }
    }
}

void write_to_jpeg(char *filename, int quality, xcb_image_t *image) {
    uint8_t data[image->width*image->height*4];
    get_rgba_image_data2(image, data);
    struct jpeg_compress_struct cinfo;
    struct jpeg_error_mgr jerr;
    FILE *outfile;
    JSAMPROW row_pointer[1];
    int row_stride;
    cinfo.err = jpeg_std_error(&jerr);
    jpeg_create_compress(&cinfo);
    if ((outfile = fopen(filename, "wb")) == NULL) {
        fprintf(stderr, "can't open %s\n", filename);
        exit(1);
    }
    jpeg_stdio_dest(&cinfo, outfile);

    cinfo.image_width = image->width;
    cinfo.image_height = image->height;
    cinfo.input_components = 4;
    cinfo.in_color_space = getJCS_EXT_RGBA();
    if (cinfo.in_color_space == JCS_UNKNOWN) {
        fprintf(stderr, "JCS_EXT_RGBA is not supported (probably built without libjpeg-trubo)");
        exit(1);
    }

    jpeg_set_defaults(&cinfo);
    jpeg_set_quality(&cinfo, quality, TRUE);

    jpeg_start_compress(&cinfo, TRUE);

    row_stride = image->width * 4;

    while (cinfo.next_scanline < cinfo.image_height) {
        row_pointer[0] = &data[cinfo.next_scanline * row_stride];
        (void) jpeg_write_scanlines(&cinfo, row_pointer, 1);
    }

    jpeg_finish_compress(&cinfo);
    fclose(outfile);
    jpeg_destroy_compress(&cinfo);
}

void write_to_jpeg_buffer(FILE *stream, int quality, xcb_image_t *image) {
    uint8_t data[image->width*image->height*4];
    get_rgba_image_data2(image, data);
    struct jpeg_compress_struct cinfo;
    struct jpeg_error_mgr jerr;
    JSAMPROW row_pointer[1];
    int row_stride;
    cinfo.err = jpeg_std_error(&jerr);
    jpeg_create_compress(&cinfo);
    jpeg_stdio_dest(&cinfo, stream);

    cinfo.image_width = image->width;
    cinfo.image_height = image->height;
    cinfo.input_components = 4;
    cinfo.in_color_space = getJCS_EXT_RGBA();
    if (cinfo.in_color_space == JCS_UNKNOWN) {
        fprintf(stderr, "JCS_EXT_RGBA is not supported (probably built without libjpeg-trubo)");
        exit(1);
    }

    jpeg_set_defaults(&cinfo);
    jpeg_set_quality(&cinfo, quality, TRUE);

    jpeg_start_compress(&cinfo, TRUE);

    row_stride = image->width * 4;

    while (cinfo.next_scanline < cinfo.image_height) {
        row_pointer[0] = &data[cinfo.next_scanline * row_stride];
        (void) jpeg_write_scanlines(&cinfo, row_pointer, 1);
    }

    jpeg_finish_compress(&cinfo);
    fclose(stream);
    jpeg_destroy_compress(&cinfo);
}

void write_to_jpeg_buffer_cuda(FILE *stream, int quality, xcb_image_t *image) {
    int N = image->width*image->height;

    struct timeval t, tt, ttt, tttt;
    gettimeofday(&t, NULL);

    uint8_t *d_data;
    uint8_t  *data;

    // hipSetDevice(0);
    // uint8_t data[image->size];
    // hipHostMalloc(&data, image->size);
    hipHostAlloc(&data, image->size, hipHostMallocMapped);
    hipMemcpy(data, image->data, image->size, hipMemcpyHostToHost);
    // memcpy(data, image->data, image->size);
    hipHostGetDevicePointer(&d_data, data, 0);

    hipMalloc(&d_data, image->size);
    // hipMemcpy(d_data, image->data, image->size, hipMemcpyHostToDevice);

    gettimeofday(&tt, NULL);

    int blocksize = 1024;
    int blocksnum = (N + blocksize - 1)/blocksize;
    get_rgba_image_data3<<<blocksnum, blocksize>>>(N, d_data);
    hipDeviceSynchronize();

    gettimeofday(&ttt, NULL);

    // hipMemcpy(data, d_data, image->size, hipMemcpyDeviceToHost);

    gettimeofday(&tttt, NULL);
    printf("cuda copy h2d: %.3fs, convert use: %.3fs, copy d2h: %.3fs\n",
            ((tt.tv_sec - t.tv_sec) * 1000000 + (tt.tv_usec - t.tv_usec))/1000000.0,
            ((ttt.tv_sec - tt.tv_sec) * 1000000 + (ttt.tv_usec - tt.tv_usec))/1000000.0,
            ((tttt.tv_sec - ttt.tv_sec) * 1000000 + (tttt.tv_usec - ttt.tv_usec))/1000000.0);

    struct jpeg_compress_struct cinfo;
    struct jpeg_error_mgr jerr;
    JSAMPROW row_pointer[1];
    int row_stride;
    cinfo.err = jpeg_std_error(&jerr);
    jpeg_create_compress(&cinfo);
    jpeg_stdio_dest(&cinfo, stream);

    cinfo.image_width = image->width;
    cinfo.image_height = image->height;
    cinfo.input_components = 4;
    cinfo.in_color_space = getJCS_EXT_RGBA();
    if (cinfo.in_color_space == JCS_UNKNOWN) {
        fprintf(stderr, "JCS_EXT_RGBA is not supported (probably built without libjpeg-trubo)");
        exit(1);
    }

    jpeg_set_defaults(&cinfo);
    jpeg_set_quality(&cinfo, quality, TRUE);

    jpeg_start_compress(&cinfo, TRUE);

    row_stride = image->width * 4;

    while (cinfo.next_scanline < cinfo.image_height) {
        row_pointer[0] = &data[cinfo.next_scanline * row_stride];
        (void) jpeg_write_scanlines(&cinfo, row_pointer, 1);
    }

    jpeg_finish_compress(&cinfo);
    fclose(stream);
    jpeg_destroy_compress(&cinfo);
    hipFree(d_data);
    hipHostFree(data);
}